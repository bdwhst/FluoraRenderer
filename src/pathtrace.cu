#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "randomUtils.h"
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "media.h"
//#include "materials.h"







__device__ inline bool util_math_is_nan(const glm::vec3& v)
{
	return (v.x != v.x) || (v.y != v.y) || (v.z != v.z);
}



//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, RGBFilm* dev_film) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = dev_film->get_image()[index];

		glm::vec3 color;
#if TONEMAPPING
		color = pix / (float)iter;
		color = max(color, glm::vec3(0.0f));
		color = util_postprocess_ACESFilm(color);
		color = color * 255.0f;
#else
		color = pix / (float)iter;
		float r = color.r, g = color.g, b = color.b;
		color = glm::clamp(glm::vec3(r, g, b) * 255.0f, glm::vec3(0.0f), glm::vec3(255.0f));

#endif
		if (util_math_is_nan(pix))
		{
			pbo[index].x = 255;
			pbo[index].y = 192;
			pbo[index].z = 203;
		}
		else
		{
			// Each thread writes one pixel location in the texture (textel)
			pbo[index].x = color.x;
			pbo[index].y = color.y;
			pbo[index].z = color.z;
		}
		pbo[index].w = 0;
	}
}


static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Object* dev_objs = NULL;
static MaterialPtr* dev_materials = NULL;
static MediumPtr* dev_media = NULL;
static MTBVHGPUNode* dev_mtbvhArray = NULL;
static Primitive* dev_primitives = NULL;
static glm::ivec3* dev_triangles = NULL;
static glm::vec3* dev_vertices = NULL;
static glm::vec2* dev_uvs = NULL;
static glm::vec3* dev_normals = NULL;
static glm::vec3* dev_tangents = NULL;
static float* dev_fsigns = NULL;
static Primitive* dev_lights = NULL;
static PathSegment* dev_paths1 = NULL;
static PathSegment* dev_paths2 = NULL;
static ShadeableIntersection* dev_intersections1 = NULL;
static ShadeableIntersection* dev_intersections2 = NULL;

static PixelSensor* dev_pixelSensor = NULL;
static RGBFilm* dev_film = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene, Allocator alloc) {
	hst_scene = scene;

	dev_pixelSensor = alloc.new_object<PixelSensor>(RGBColorSpace::sRGB, nullptr, 0.03, alloc);

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	dev_film = alloc.new_object<RGBFilm>(dev_image, RGBColorSpace::sRGB, 100.0f);

	hipMalloc(&dev_paths1, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_paths2, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_objs, scene->objects.size() * sizeof(Object));
	hipMemcpy(dev_objs, scene->objects.data(), scene->objects.size() * sizeof(Object), hipMemcpyHostToDevice);

	if (scene->triangles.size())
	{
		hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(glm::ivec3));
		hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(glm::ivec3), hipMemcpyHostToDevice);

		hipMalloc(&dev_vertices, scene->verticies.size() * sizeof(glm::vec3));
		hipMemcpy(dev_vertices, scene->verticies.data(), scene->verticies.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

		hipMalloc(&dev_uvs, scene->uvs.size() * sizeof(glm::vec2));
		hipMemcpy(dev_uvs, scene->uvs.data(), scene->uvs.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);
		if (scene->normals.size())
		{
			hipMalloc(&dev_normals, scene->normals.size() * sizeof(glm::vec3));
			hipMemcpy(dev_normals, scene->normals.data(), scene->normals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
		}
		if (scene->tangents.size())
		{
			hipMalloc(&dev_tangents, scene->tangents.size() * sizeof(glm::vec3));
			hipMemcpy(dev_tangents, scene->tangents.data(), scene->tangents.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
		}
		if (scene->fSigns.size())
		{
			hipMalloc(&dev_fsigns, scene->fSigns.size() * sizeof(float));
			hipMemcpy(dev_fsigns, scene->fSigns.data(), scene->fSigns.size() * sizeof(float), hipMemcpyHostToDevice);
		}
	}

#if MTBVH
	hipMalloc(&dev_mtbvhArray, scene->MTBVHArray.size() * sizeof(MTBVHGPUNode));
	hipMemcpy(dev_mtbvhArray, scene->MTBVHArray.data(), scene->MTBVHArray.size() * sizeof(MTBVHGPUNode), hipMemcpyHostToDevice);
#else
	hipMalloc(&dev_bvhArray, scene->bvhArray.size() * sizeof(BVHGPUNode));
	hipMemcpy(dev_bvhArray, scene->bvhArray.data(), scene->bvhArray.size() * sizeof(BVHGPUNode), hipMemcpyHostToDevice);
#endif

	hipMalloc(&dev_primitives, scene->primitives.size() * sizeof(Primitive));
	hipMemcpy(dev_primitives, scene->primitives.data(), scene->primitives.size() * sizeof(Primitive), hipMemcpyHostToDevice);

	if (scene->lights.size())
	{
		hipMalloc(&dev_lights, scene->lights.size() * sizeof(Primitive));
		hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Primitive), hipMemcpyHostToDevice);
	}

	if (scene->materials.size())
	{
		hipMalloc(&dev_materials, scene->materials.size() * sizeof(MaterialPtr));
		hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(MaterialPtr), hipMemcpyHostToDevice);
	}
	
	if (scene->media.size())
	{
		hipMalloc(&dev_media, scene->media.size() * sizeof(MediumPtr));
		hipMemcpy(dev_media, scene->media.data(), scene->media.size() * sizeof(MediumPtr), hipMemcpyHostToDevice);
	}


	hipMalloc(&dev_intersections1, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections1, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_intersections2, pixelcount * sizeof(ShadeableIntersection));


#if !STOCHASTIC_SAMPLING && FIRST_INTERSECTION_CACHING
	hipMalloc(&dev_intersectionCache, pixelcount * sizeof(ShadeableIntersection));
	hipMalloc(&dev_pathCache, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_rayValidCache, pixelcount * sizeof(int));
	hipMalloc(&dev_imageCache, pixelcount * sizeof(glm::vec3));
#endif
	// TODO: initialize any extra device memeory you need

	checkCUDAError("pathtraceInit");
}

void pathtraceFree(Scene* scene) {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths1);
	hipFree(dev_paths2);
	hipFree(dev_objs);
	if (scene->triangles.size())
	{
		hipFree(dev_triangles);
		hipFree(dev_vertices);
		hipFree(dev_uvs);
		if (scene->normals.size())
		{
			hipFree(dev_normals);
		}
		if (scene->tangents.size())
		{
			hipFree(dev_tangents);
		}
		if (scene->fSigns.size())
		{
			hipFree(dev_fsigns);
		}
	}
	hipFree(dev_primitives);
	if (scene->lights.size())
	{
		hipMalloc(&dev_lights, scene->lights.size() * sizeof(Primitive));
		hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Primitive), hipMemcpyHostToDevice);
	}
#if MTBVH
	hipFree(dev_mtbvhArray);
#else
	hipFree(dev_bvhArray);
#endif
	hipFree(dev_materials);
	hipFree(dev_media);
	hipFree(dev_intersections1);
	hipFree(dev_intersections2);
#if !STOCHASTIC_SAMPLING && FIRST_INTERSECTION_CACHING
	hipFree(dev_intersectionCache);
	hipFree(dev_pathCache);
	hipFree(dev_rayValidCache);
	hipFree(dev_imageCache);
#endif
	// TODO: clean up any extra device memory you created

	checkCUDAError("pathtraceFree");
}

__device__ inline glm::vec2 util_concentric_sample_disk(glm::vec2 rand)
{
	rand = 2.0f * rand - 1.0f;
	if (rand.x == 0 && rand.y == 0)
	{
		return glm::vec2(0);
	}
	const float pi_4 = PI / 4, pi_2 = PI / 2;
	bool x_g_y = abs(rand.x) > abs(rand.y);
	float theta = x_g_y ? pi_4 * rand.y / rand.x : pi_2 - pi_4 * rand.x / rand.y;
	float r = x_g_y ? rand.x : rand.y;
	return glm::vec2(cos(theta), sin(theta)) * r;
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	thrust::default_random_engine rng = makeSeededRandomEngine(iter, x * cam.resolution.y + y, 0);
	thrust::uniform_real_distribution<float> u01(0, 1);

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.transport = SampledSpectrum(1.0f);
		segment.lambda = SampledWavelengths::sample_visible(u01(rng));
		//segment.lambda = SampledWavelengths::sample_uniform(u01(rng));
#if STOCHASTIC_SAMPLING
		glm::vec2 jitter = glm::vec2(0.5f * (u01(rng) * 2.0f - 1.0f), 0.5f * (u01(rng) * 2.0f - 1.0f));
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + jitter[0])
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + jitter[1])
		);
#if DOF_ENABLED
		float lensR = cam.lensRadius;
		glm::vec3 perpDir = glm::cross(cam.right, cam.up);
		perpDir = glm::normalize(perpDir);
		float focalLen = cam.focalLength;
		float tFocus = focalLen / glm::abs(glm::dot(segment.ray.direction, perpDir));
		glm::vec2 offset = lensR * util_concentric_sample_disk(glm::vec2(u01(rng), u01(rng)));
		glm::vec3 newOri = offset.x * cam.right + offset.y * cam.up + cam.position;
		glm::vec3 pFocus = segment.ray.direction * tFocus + segment.ray.origin;
		segment.ray.direction = glm::normalize(pFocus - newOri);
		segment.ray.origin = newOri;
#endif

#else
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);
#endif
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
		segment.lastMatPdf = -1;
		// TODO: change this to camera's medium
		segment.ray.medium = -1;
		segment.rng = rng;
	}
}



__global__ void compute_intersection_bvh_no_volume(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, SceneInfoDev dev_sceneInfo
	, ShadeableIntersection* intersections
	, int* rayValid
	, RGBFilm* dev_film
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index >= num_paths) return;
	PathSegment& pathSegment = pathSegments[path_index];
	Ray& ray = pathSegment.ray;
	glm::vec3 rayDir = pathSegment.ray.direction;
	glm::vec3 rayOri = pathSegment.ray.origin;
	float x = fabs(rayDir.x), y = fabs(rayDir.y), z = fabs(rayDir.z);
	int axis = x > y && x > z ? 0 : (y > z ? 1 : 2);
	int sgn = rayDir[axis] > 0 ? 0 : 1;
	int d = (axis << 1) + sgn;
	const MTBVHGPUNode* currArray = dev_sceneInfo.dev_mtbvhArray + d * dev_sceneInfo.bvhDataSize;
	int curr = 0;
	ShadeableIntersection tmpIntersection;
	tmpIntersection.t = FLT_MAX;
	bool intersected = false;
	while (curr >= 0 && curr < dev_sceneInfo.bvhDataSize)
	{
		bool outside = true;
		float boxt = boundingBoxIntersectionTest(currArray[curr].bbox, ray, outside);
		if (!outside) boxt = EPSILON;
		if (boxt > 0 && boxt < tmpIntersection.t)
		{
			if (currArray[curr].startPrim != -1)//leaf node
			{
				int start = currArray[curr].startPrim, end = currArray[curr].endPrim;
				bool intersect = util_bvh_leaf_intersect(start, end, dev_sceneInfo, &ray, &tmpIntersection);
				intersected = intersected || intersect;
			}
			curr = currArray[curr].hitLink;
		}
		else
		{
			curr = currArray[curr].missLink;
		}
	}
	
	rayValid[path_index] = intersected;
	if (intersected)
	{
		intersections[path_index] = tmpIntersection;
		pathSegment.remainingBounces--;
	}
	else if (dev_sceneInfo.skyboxObj)
	{
		glm::vec2 uv = util_sample_spherical_map(glm::normalize(rayDir));
		float4 skyColorRGBA = tex2D<float4>(dev_sceneInfo.skyboxObj, uv.x, uv.y);
#if WHITE_FURNANCE_TEST
		glm::vec3 skyColor = glm::vec3(1.0, 1.0, 1.0);
#else
		glm::vec3 skyColor = glm::vec3(skyColorRGBA.x, skyColorRGBA.y, skyColorRGBA.z);
#endif
		const RGBColorSpace* colorSpace = RGBColorSpace_sRGB;
		RGBIlluminantSpectrum illumSpec(*colorSpace, skyColor);
		SampledSpectrum skyRadiance = illumSpec.sample(pathSegment.lambda);
		glm::vec3 sensorRGB = dev_sceneInfo.pixelSensor->to_sensor_rgb(pathSegment.transport * skyRadiance, pathSegment.lambda);
		dev_film->add_radiance(sensorRGB, pathSegment.pixelIndex);
	}
}


// Does not handle surface intersection
__global__ void compute_intersection_bvh_volume_naive(
	int iter
	, int depth
	, int num_paths
	, PathSegment* pathSegments
	, SceneInfoDev dev_sceneInfo
	, ShadeableIntersection* intersections
	, int* rayValid
	, RGBFilm* dev_film
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index >= num_paths) return;
	PathSegment& pathSegment = pathSegments[path_index];
	Ray& ray = pathSegment.ray;
	glm::vec3 rayDir = pathSegment.ray.direction;
	glm::vec3 rayOri = pathSegment.ray.origin;
	float x = fabs(rayDir.x), y = fabs(rayDir.y), z = fabs(rayDir.z);
	int axis = x > y && x > z ? 0 : (y > z ? 1 : 2);
	int sgn = rayDir[axis] > 0 ? 0 : 1;
	int d = (axis << 1) + sgn;
	const MTBVHGPUNode* currArray = dev_sceneInfo.dev_mtbvhArray + d * dev_sceneInfo.bvhDataSize;
	int curr = 0;
	ShadeableIntersection tmpIntersection;
	tmpIntersection.t = FLT_MAX;
	tmpIntersection.materialId = -1;
	bool intersected_surface = false;
	while (curr >= 0 && curr < dev_sceneInfo.bvhDataSize)
	{
		bool outside = true;
		float boxt = boundingBoxIntersectionTest(currArray[curr].bbox, ray, outside);
		if (!outside) boxt = EPSILON;
		if (boxt > 0 && boxt < tmpIntersection.t)
		{
			if (currArray[curr].startPrim != -1)//leaf node
			{
				int start = currArray[curr].startPrim, end = currArray[curr].endPrim;
				bool intersect = util_bvh_leaf_intersect(start, end, dev_sceneInfo, &ray, &tmpIntersection);
				intersected_surface = intersected_surface || intersect;
			}
			curr = currArray[curr].hitLink;
		}
		else
		{
			curr = currArray[curr].missLink;
		}
	}
	pathSegment.lambda.terminate_secondary();
	bool scattered_in_medium = false, absorbed_in_medium = false;

	// FIXED: Triangle intersection is now watertight
	if (intersected_surface && depth == 0 && ray.medium != -1)
	{
		assert(0);
	}

	if (ray.medium != -1)
	{
		thrust::default_random_engine& rng = pathSegment.rng;
		thrust::uniform_int_distribution<int> int_dist;
		thrust::default_random_engine tmaj_rng(int_dist(rng));

		float t_max = intersected_surface ? tmpIntersection.t : FLT_MAX;
		sample_Tmaj(dev_sceneInfo.dev_media, ray, t_max, tmaj_rng, pathSegment.lambda, [&](const glm::vec3& p, MediumProperties mp, SampledSpectrum sigma_maj, SampledSpectrum Tmaj) {
			float pAbsorb = mp.sigma_a[0] / sigma_maj[0];
			float pScatter = mp.sigma_s[0] / sigma_maj[0];
			float pNull = math::max(0.0f, 1 - pAbsorb - pScatter);
			if (pNull == 1.0f)
			{
				return true;
			}
			thrust::uniform_real_distribution<float> u01(0, 1);
			float uMode = u01(tmaj_rng);
			if (uMode < pAbsorb)
			{
				glm::vec3 sensorRGB = dev_sceneInfo.pixelSensor->to_sensor_rgb(pathSegment.transport * mp.Le, pathSegment.lambda);
				dev_film->add_radiance(sensorRGB, pathSegment.pixelIndex);
				absorbed_in_medium = true;
				return false;
			}
			else if (uMode >= pAbsorb && uMode < pAbsorb + pScatter)
			{
				pathSegment.remainingBounces--;
				int bounces = pathSegment.remainingBounces;
				if (bounces == 0)
				{
					return false;
				}

				glm::vec2 u(u01(tmaj_rng), u01(tmaj_rng));
				glm::vec3 wi;
				float pdf = 0.0f;
				float phase = mp.phase.sample_p(-ray.direction, u, &wi, &pdf);
				if (pdf == 0) 
				{
					return false;
				}
				ray.origin = p;
				assert((wi.x != 0.0f) || (wi.y != 0.0f) || (wi.z != 0.0f));
				ray.direction = wi;
				
				pathSegment.transport *= phase / pdf;
				scattered_in_medium = true;
				return false;
			}
			else
			{
				return true;
			}
			});
	}
	if (absorbed_in_medium)
	{
		rayValid[path_index] = false;
		return;
	}

	// If real scatter occurs, mark materialId as -1
	if (scattered_in_medium)
	{
		intersections[path_index].materialId = -1;
		rayValid[path_index] = true;
		return;
	}
	// If there is no real scatter and a intersection with surface occurs
	// We are intersecting with a medium interface or a light surface
	// Continue travese through the current ray dir, but change the origin to be the intersection point
	if (intersected_surface)
	{
		intersections[path_index] = tmpIntersection;
		ray.origin = tmpIntersection.worldPos + ray.direction * SCATTER_ORIGIN_OFFSETMULT;
		rayValid[path_index] = true;
		return;
	}
	// If there is no scatter in media and intersection with surface
	// Try to read the radiance from skybox
	if (dev_sceneInfo.skyboxObj)
	{
		glm::vec2 uv = util_sample_spherical_map(glm::normalize(rayDir));
		float4 skyColorRGBA = tex2D<float4>(dev_sceneInfo.skyboxObj, uv.x, uv.y);
#if WHITE_FURNANCE_TEST
		glm::vec3 skyColor = glm::vec3(1.0, 1.0, 1.0);
#else
		glm::vec3 skyColor = glm::vec3(skyColorRGBA.x, skyColorRGBA.y, skyColorRGBA.z);
#endif
		const RGBColorSpace* colorSpace = RGBColorSpace_sRGB;
		RGBIlluminantSpectrum illumSpec(*colorSpace, skyColor);
		SampledSpectrum skyRadiance = illumSpec.sample(pathSegment.lambda);
		glm::vec3 sensorRGB = dev_sceneInfo.pixelSensor->to_sensor_rgb(pathSegment.transport * skyRadiance, pathSegment.lambda);
		dev_film->add_radiance(sensorRGB, pathSegment.pixelIndex);
		rayValid[path_index] = false;
	}
}

//__global__ void draw_gbuffer(
//	int num_paths
//	, PathSegment* pathSegments
//	, SceneInfoDev dev_sceneInfo
//	, SceneGbuffer gbuffer
//)
//{
//	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
//	if (path_index >= num_paths) return;
//	PathSegment& pathSegment = pathSegments[path_index];
//	Ray& ray = pathSegment.ray;
//	glm::vec3 rayDir = pathSegment.ray.direction;
//	glm::vec3 rayOri = pathSegment.ray.origin;
//	float x = fabs(rayDir.x), y = fabs(rayDir.y), z = fabs(rayDir.z);
//	int axis = x > y && x > z ? 0 : (y > z ? 1 : 2);
//	int sgn = rayDir[axis] > 0 ? 0 : 1;
//	int d = (axis << 1) + sgn;
//	const MTBVHGPUNode* currArray = dev_sceneInfo.dev_mtbvhArray + d * dev_sceneInfo.bvhDataSize;
//	int curr = 0;
//	ShadeableIntersection tmpIntersection;
//	tmpIntersection.t = 1e37f;
//	bool intersected = false;
//	while (curr >= 0 && curr < dev_sceneInfo.bvhDataSize)
//	{
//		bool outside = true;
//		float boxt = boundingBoxIntersectionTest(currArray[curr].bbox, ray, outside);
//		if (!outside) boxt = EPSILON;
//		if (boxt > 0 && boxt < tmpIntersection.t)
//		{
//			if (currArray[curr].startPrim != -1)//leaf node
//			{
//				int start = currArray[curr].startPrim, end = currArray[curr].endPrim;
//				bool intersect = util_bvh_leaf_intersect(start, end, dev_sceneInfo, ray, &tmpIntersection);
//				intersected = intersected || intersect;
//			}
//			curr = currArray[curr].hitLink;
//		}
//		else
//		{
//			curr = currArray[curr].missLink;
//		}
//	}
//	if (intersected)
//	{
//		int pixelIdx = pathSegment.pixelIndex;
//		gbuffer.dev_normal[pixelIdx] += tmpIntersection.surfaceNormal;
//		Material& mat = dev_sceneInfo.dev_materials[tmpIntersection.materialId];
//		glm::vec3 materialColor = mat.color;
//		if (mat.baseColorMap)
//		{
//			float4 color = tex2D<float4>(mat.baseColorMap, tmpIntersection.uv.x, tmpIntersection.uv.y);
//			materialColor.x = color.x;
//			materialColor.y = color.y;
//			materialColor.z = color.z;
//		}
//		gbuffer.dev_albedo[pixelIdx] += materialColor;
//	}
//	else
//	{
//		if (dev_sceneInfo.skyboxObj)
//		{
//			glm::vec2 uv = util_sample_spherical_map(glm::normalize(rayDir));
//			float4 skyColorRGBA = tex2D<float4>(dev_sceneInfo.skyboxObj, uv.x, uv.y);
//			glm::vec3 skyColor = glm::vec3(skyColorRGBA.x, skyColorRGBA.y, skyColorRGBA.z);
//			gbuffer.dev_albedo[pathSegment.pixelIndex] += skyColor;
//		}
//	}
//}


__global__ void scatter_on_intersection(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, SceneInfoDev sceneInfo
	, int* rayValid
	, RGBFilm* dev_film
)
{
	extern __shared__ char sharedMemory[];
	char* bxdfBufferLocal = sharedMemory;

	MaterialPtr* materials = sceneInfo.dev_materials;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths) return;
	ShadeableIntersection intersection = shadeableIntersections[idx];
	// Set up the RNG
	// LOOK: this is how you use thrust's RNG! Please look at
	// makeSeededRandomEngine as well.
	thrust::default_random_engine& rng = pathSegments[idx].rng;
	thrust::uniform_real_distribution<float> u01(0, 1);
	// scattered in media
	if (intersection.materialId == -1)
	{
		rayValid[idx] = 1;
		return;
	}
	MaterialPtr material = materials[intersection.materialId];

//#if VIS_NORMAL
//	image[pathSegments[idx].pixelIndex] += (glm::normalize(intersection.surfaceNormal));
//	rayValid[idx] = 0;
//	return;
//#endif

	// If the material indicates that the object was a light, "light" the ray
	if (material.Is<EmissiveMaterial>()) {
		pathSegments[idx].transport *= material.Cast<EmissiveMaterial>()->Le(pathSegments[idx].lambda);
		rayValid[idx] = 0;
		if (!pathSegments[idx].transport.is_nan())
		{
			dev_film->add_radiance(sceneInfo.pixelSensor->to_sensor_rgb(pathSegments[idx].transport, pathSegments[idx].lambda), pathSegments[idx].pixelIndex);
		}
	}
	else {
		// For now if we encounter some non-emissive surface while rendering volumetrics, just error exit
		assert(sceneInfo.containsVolume == false);
		glm::vec3& woInWorld = pathSegments[idx].ray.direction;
		glm::vec3 nMap = glm::vec3(0, 0, 1);
		

		//if (material.normalMap != 0)
		//{
		//	float4 nMapCol = tex2D<float4>(material.normalMap, intersection.uv.x, intersection.uv.y);
		//	nMap.x = nMapCol.x;
		//	nMap.y = nMapCol.y;
		//	nMap.z = nMapCol.z;
		//	nMap = glm::pow(nMap, glm::vec3(1 / 2.2f));
		//	nMap = nMap * 2.0f - 1.0f;
		//	nMap = glm::normalize(nMap);
		//}
		glm::vec3 N = glm::normalize(intersection.surfaceNormal);
		glm::vec3 B, T;
		//if (material.normalMap != 0)
		//{
		//	T = intersection.surfaceTangent;
		//	T = glm::normalize(T - N * glm::dot(N, T));
		//	B = glm::cross(N, T);
		//	N = glm::normalize(T * nMap.x + B * nMap.y + N * nMap.z);
		//}
		//else
		//{
		math::Frame frame = math::Frame::from_z(N);
		//}
		glm::vec3 wo = frame.to_local(-woInWorld);
		wo = glm::normalize(wo);
		float pdf = 0;
		glm::vec3 wi;

		MaterialEvalInfo info(wo, intersection.uv, pathSegments[idx].lambda);

		BxDFPtr bxdf = material.get_bxdf(info, bxdfBufferLocal + threadIdx.x * BxDFMaxSize);

		SampledSpectrum f = bxdf.sample_f(wo, wi, pdf, rng);

		//glm::vec3 wi, bxdf;
		//glm::vec3 random = glm::vec3(u01(rng), u01(rng), u01(rng));
		//float cosWi = 0;
		//if (material.type == MaterialType::metallicWorkflow)
		//{
		//	float4 color = { 0,0,0,1 };
		//	float roughness = material.roughness, metallic = material.metallic;
		//	if (material.baseColorMap != 0)
		//	{
		//		color = tex2D<float4>(material.baseColorMap, intersection.uv.x, intersection.uv.y);
		//		materialColor.x = color.x;
		//		materialColor.y = color.y;
		//		materialColor.z = color.z;
		//	}
		//	if (material.metallicRoughnessMap != 0)
		//	{
		//		color = tex2D<float4>(material.metallicRoughnessMap, intersection.uv.x, intersection.uv.y);
		//		roughness = color.y;
		//		metallic = color.z;
		//	}

		//	bxdf = bxdf_metallic_workflow_sample_f(wo, &wi, random, &pdf, materialColor, metallic, roughness);
		//	cosWi = util_math_tangent_space_clampedcos(wi);
		//}
		//else if (material.type == MaterialType::frenselSpecular)
		//{
		//	glm::vec2 iors = glm::dot(woInWorld, N) < 0 ? glm::vec2(1.0, material.indexOfRefraction) : glm::vec2(material.indexOfRefraction, 1.0);
		//	bxdf = bxdf_frensel_specular_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor, materialColor, iors);
		//	cosWi = 1.0;
		//}
		//else if (material.type == MaterialType::microfacet)
		//{
		//	bxdf = bxdf_microfacet_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor, material.roughness);
		//	cosWi = util_math_tangent_space_clampedcos(wi);
		//}
		//else if (material.type == MaterialType::blinnphong)
		//{
		//	bxdf = bxdf_blinn_phong_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor, material.specExponent);
		//	cosWi = util_math_tangent_space_clampedcos(wi);
		//}
		//else if (material.type == MaterialType::asymMicrofacet)
		//{
		//	if(material.asymmicrofacet.type == conductor)
		//		bxdf = bxdf_asymConductor_sample_f(wo, &wi, rng, &pdf, material.asymmicrofacet, NUM_MULTI_SCATTER_BOUNCE);
		//	else
		//		bxdf = bxdf_asymDielectric_sample_f(wo, &wi, rng, &pdf, material.asymmicrofacet, NUM_MULTI_SCATTER_BOUNCE);
		//	cosWi = 1.0f;
		//}
		//else//diffuse
		//{
		//	float4 color = { 0,0,0,1 };
		//	if (material.baseColorMap != 0)
		//	{
		//		color = tex2D<float4>(material.baseColorMap, intersection.uv.x, intersection.uv.y);
		//		materialColor.x = color.x;
		//		materialColor.y = color.y;
		//		materialColor.z = color.z;
		//	}
		//	
		//	if (color.w <= ALPHA_CUTOFF)
		//	{
		//		bxdf = pathSegments[idx].remainingBounces == 0 ? glm::vec3(0, 0, 0) : glm::vec3(1, 1, 1);
		//		wi = -wo;
		//		pdf = abs(wi.z);
		//	}
		//	else
		//	{
		//		bxdf = bxdf_diffuse_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor);
		//	}
		//	cosWi = abs(wi.z);

		//}
		if (pdf > 0)
		{
			pathSegments[idx].transport *= f / pdf;
			glm::vec3 newDir = glm::normalize(frame.from_local(wi));
			glm::vec3 offset = glm::dot(newDir, N) < 0 ? -N : N;
			float offsetMult = !material.Is<DielectricMaterial>() ? SCATTER_ORIGIN_OFFSETMULT : SCATTER_ORIGIN_OFFSETMULT * 100.0f;
			pathSegments[idx].ray.origin = intersection.worldPos + offset * offsetMult;
			pathSegments[idx].ray.direction = newDir;
			rayValid[idx] = 1;
		}
		else
		{
			rayValid[idx] = 0;
		}

	}
}

//__global__ void scatter_on_intersection_mis(
//	int iter
//	, int num_paths
//	, ShadeableIntersection* shadeableIntersections
//	, PathSegment* pathSegments
//	, SceneInfoDev sceneInfo
//	, int* rayValid
//	, glm::vec3* image
//)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	if (idx >= num_paths) return;
//	ShadeableIntersection intersection = shadeableIntersections[idx];
//	thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
//	thrust::uniform_real_distribution<float> u01(0, 1);
//
//	Material* materials = sceneInfo.dev_materials;
//	Material material = materials[intersection.materialId];
//	glm::vec3 materialColor = material.color;
//#if VIS_NORMAL
//	image[pathSegments[idx].pixelIndex] += (glm::normalize(intersection.surfaceNormal));
//	rayValid[idx] = 0;
//	return;
//#endif
//
//	// If the material indicates that the object was a light, "light" the ray
//	if (material.type == MaterialType::emitting) {
//		int lightPrimId = intersection.primitiveId;
//		
//		float matPdf = pathSegments[idx].lastMatPdf;
//		if (matPdf > 0.0)
//		{
//			float G = util_math_solid_angle_to_area(intersection.worldPos, intersection.surfaceNormal, pathSegments[idx].ray.origin);
//			//We do not know the value of light pdf(of last intersection point) of the sample taken from bsdf sampling unless we hit a light
//			float lightPdf = lights_sample_pdf(sceneInfo, lightPrimId);
//			//Computing weights from last intersection point
//			float misW = util_mis_weight(matPdf * G, lightPdf);
//			pathSegments[idx].transport *= (materialColor * material.emittance * misW);
//		}
//		else//This ray hits a light directly
//		{
//			pathSegments[idx].transport *= (materialColor * material.emittance);
//		}
//		rayValid[idx] = 0;
//		if (!util_math_is_nan(pathSegments[idx].transport))
//			image[pathSegments[idx].pixelIndex] += pathSegments[idx].transport;
//	}
//	else {
//		//Prepare normal and wo for sample
//		glm::vec3& woInWorld = pathSegments[idx].ray.direction;
//		glm::vec3 nMap = glm::vec3(0, 0, 1);
//		if (material.normalMap != 0)
//		{
//			float4 nMapCol = tex2D<float4>(material.normalMap, intersection.uv.x, intersection.uv.y);
//			nMap.x = nMapCol.x;
//			nMap.y = nMapCol.y;
//			nMap.z = nMapCol.z;
//			nMap = glm::pow(nMap, glm::vec3(1 / 2.2f));
//			nMap = nMap * 2.0f - 1.0f;
//			nMap = glm::normalize(nMap);
//		}
//		glm::vec3 N = glm::normalize(intersection.surfaceNormal);
//		glm::vec3 B, T;
//		if (material.normalMap != 0)
//		{
//			T = intersection.surfaceTangent;
//			T = glm::normalize(T - N * glm::dot(N, T));
//			B = glm::cross(N, T);
//			N = glm::normalize(T * nMap.x + B * nMap.y + N * nMap.z);
//		}
//		else
//		{
//			util_math_get_TBN_pixar(N, &T, &B);
//		}
//		glm::mat3 TBN(T, B, N);
//		glm::vec3 wo = glm::transpose(TBN) * (-woInWorld);
//		wo = glm::normalize(wo);
//		float pdf = 0;
//		glm::vec3 wi, bxdf;
//		glm::vec3 random = glm::vec3(u01(rng), u01(rng), u01(rng));
//		float cosWi = 0;
//		if (material.type == MaterialType::frenselSpecular)
//		{
//			glm::vec2 iors = glm::dot(woInWorld, N) < 0 ? glm::vec2(1.0, material.indexOfRefraction) : glm::vec2(material.indexOfRefraction, 1.0);
//			bxdf = bxdf_frensel_specular_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor, materialColor, iors);
//			cosWi = 1.0;
//		}
//		else
//		{
//			float roughness = material.roughness, metallic = material.metallic;
//			float specExp = material.specExponent;
//			float4 color = { 0,0,0,1 };
//			float alpha = 1.0f;
//			//Texture mapping
//			if (material.baseColorMap != 0)
//			{
//				color = tex2D<float4>(material.baseColorMap, intersection.uv.x, intersection.uv.y);
//				materialColor.x = color.x;
//				materialColor.y = color.y;
//				materialColor.z = color.z;
//				alpha = color.w;
//			}
//			if (material.metallicRoughnessMap != 0)
//			{
//				color = tex2D<float4>(material.metallicRoughnessMap, intersection.uv.x, intersection.uv.y);
//				roughness = color.y;
//				metallic = color.z;
//			}
//			//Sampling lights
//			glm::vec3 lightPos, lightNormal, emissive = glm::vec3(0);
//			float light_pdf = -1.0;
//			glm::vec3 offseted_pos = intersection.worldPos + N * SCATTER_ORIGIN_OFFSETMULT;
//			lights_sample(sceneInfo, glm::vec3(u01(rng), u01(rng), u01(rng)), offseted_pos, N, &lightPos, &lightNormal, &emissive, &light_pdf);
//			glm::vec3 light_bxdf = glm::vec3(0);
//			
//			if (emissive.x > 0.0 || emissive.y > 0.0 || emissive.z > 0.0)
//			{
//				glm::vec3 light_wi = lightPos - offseted_pos;
//				light_wi = glm::normalize(glm::transpose(TBN) * (light_wi));
//				float G = util_math_solid_angle_to_area(lightPos, lightNormal, offseted_pos);
//				float mat_pdf = -1.0f;
//				if (material.type == MaterialType::metallicWorkflow)
//				{
//					mat_pdf = bxdf_metallic_workflow_pdf(wo, light_wi, materialColor, metallic, roughness);
//					light_bxdf = bxdf_metallic_workflow_eval(wo, light_wi, materialColor, metallic, roughness);
//				}
//				else if (material.type == MaterialType::microfacet)
//				{
//					mat_pdf = bxdf_microfacet_pdf(wo, light_wi, roughness);
//					light_bxdf = bxdf_microfacet_eval(wo, light_wi, materialColor, roughness);
//				}
//				else if (material.type == MaterialType::blinnphong)
//				{
//					mat_pdf = bxdf_blinn_phong_pdf(wo, light_wi, specExp);
//					light_bxdf = bxdf_blinn_phong_eval(wo, light_wi, materialColor, specExp);
//				}
//				else
//				{
//					mat_pdf = bxdf_diffuse_pdf(wo, light_wi);
//					light_bxdf = bxdf_diffuse_eval(wo, light_wi, materialColor);
//				}
//				float misW = util_mis_weight(light_pdf, mat_pdf * G);
//				image[pathSegments[idx].pixelIndex] += pathSegments[idx].transport * light_bxdf * util_math_tangent_space_clampedcos(light_wi) * emissive * misW * G / light_pdf;
//			}
//			//Sampling material bsdf
//			if (material.type == MaterialType::metallicWorkflow)
//			{	
//				bxdf = bxdf_metallic_workflow_sample_f(wo, &wi, random, &pdf, materialColor, metallic, roughness);
//			}
//			else if (material.type == MaterialType::microfacet)
//			{
//				bxdf = bxdf_microfacet_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor, roughness);
//			}
//			else if (material.type == MaterialType::blinnphong)
//			{
//				bxdf = bxdf_blinn_phong_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor, specExp);
//			}
//			else//diffuse
//			{
//				if (alpha <= ALPHA_CUTOFF)
//				{
//					bxdf = pathSegments[idx].remainingBounces == 0 ? glm::vec3(0, 0, 0) : glm::vec3(1, 1, 1);
//					wi = -wo;
//					pdf = util_math_tangent_space_clampedcos(wi);
//				}
//				else
//				{
//					bxdf = bxdf_diffuse_sample_f(wo, &wi, glm::vec2(random.x, random.y), &pdf, materialColor);
//				}
//
//			}
//			cosWi = util_math_tangent_space_clampedcos(wi);
//		}
//		if (pdf > 0)
//		{
//			pathSegments[idx].transport *= bxdf * cosWi / pdf;
//			glm::vec3 newDir = glm::normalize(TBN * wi);
//			glm::vec3 offset = glm::dot(newDir, N) < 0 ? -N : N;
//			float offsetMult = material.type != MaterialType::frenselSpecular ? SCATTER_ORIGIN_OFFSETMULT : SCATTER_ORIGIN_OFFSETMULT * 100.0f;
//			pathSegments[idx].ray.origin = intersection.worldPos + offset * offsetMult;
//			pathSegments[idx].ray.direction = newDir;
//			pathSegments[idx].lastMatPdf = pdf;
//			rayValid[idx] = 1;
//		}
//		else
//		{
//			rayValid[idx] = 0;
//		}
//
//	}
//}


//__global__ void addBackground(glm::vec3* dev_image, glm::vec3* dev_imageCache, int numPixels)
//{
//	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
//	if (index >= numPixels) return;
//	dev_image[index] += dev_imageCache[index];
//}



struct mat_comp {
	__host__ __device__ bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b) const {
		return a.type < b.type;
	}
};

int compact_rays(int* rayValid,int* rayIndex,int numRays, bool sortByMat=false)
{
	thrust::device_ptr<PathSegment> dev_thrust_paths1(dev_paths1), dev_thrust_paths2(dev_paths2);
	thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections1(dev_intersections1), dev_thrust_intersections2(dev_intersections2);
	thrust::device_ptr<int> dev_thrust_rayValid(rayValid), dev_thrust_rayIndex(rayIndex);
	thrust::exclusive_scan(dev_thrust_rayValid, dev_thrust_rayValid + numRays, dev_thrust_rayIndex);
	int nextNumRays, tmp;
	hipMemcpy(&tmp, rayIndex + numRays - 1, sizeof(int), hipMemcpyDeviceToHost);
	nextNumRays = tmp;
	hipMemcpy(&tmp, rayValid + numRays - 1, sizeof(int), hipMemcpyDeviceToHost);
	nextNumRays += tmp;
	thrust::scatter_if(dev_thrust_paths1, dev_thrust_paths1 + numRays, dev_thrust_rayIndex, dev_thrust_rayValid, dev_thrust_paths2);
	thrust::scatter_if(dev_thrust_intersections1, dev_thrust_intersections1 + numRays, dev_thrust_rayIndex, dev_thrust_rayValid, dev_thrust_intersections2);
	if (sortByMat)
	{
		mat_comp cmp;
		thrust::sort_by_key(dev_thrust_intersections2, dev_thrust_intersections2 + nextNumRays, dev_thrust_paths2, cmp);
	}
	std::swap(dev_paths1, dev_paths2);
	std::swap(dev_intersections1, dev_intersections2);
	return nextNumRays;
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing
	SceneInfoDev dev_sceneInfo{};
	dev_sceneInfo.dev_materials = dev_materials;
	if (dev_media)
	{
		dev_sceneInfo.dev_media = dev_media;
		dev_sceneInfo.containsVolume = true;
	}
	dev_sceneInfo.dev_objs = dev_objs;
	dev_sceneInfo.objectsSize = hst_scene->objects.size();
	dev_sceneInfo.modelInfo.dev_triangles = dev_triangles;
	dev_sceneInfo.modelInfo.dev_vertices = dev_vertices;
	dev_sceneInfo.modelInfo.dev_normals = dev_normals;
	dev_sceneInfo.modelInfo.dev_uvs = dev_uvs;
	dev_sceneInfo.modelInfo.dev_tangents = dev_tangents;
	dev_sceneInfo.modelInfo.dev_fsigns = dev_fsigns;
	dev_sceneInfo.dev_primitives = dev_primitives;
#if USE_BVH
#if MTBVH
	dev_sceneInfo.dev_mtbvhArray = dev_mtbvhArray;
	dev_sceneInfo.bvhDataSize = hst_scene->MTBVHArray.size() / 6;
#else
	dev_sceneInfo.dev_bvhArray = dev_bvhArray;
	dev_sceneInfo.bvhDataSize = hst_scene->bvhTreeSize;
#endif
#endif // 
	dev_sceneInfo.skyboxObj = hst_scene->skyboxTextureObj;
	dev_sceneInfo.dev_lights = dev_lights;
	dev_sceneInfo.lightsSize = hst_scene->lights.size();

	dev_sceneInfo.pixelSensor = dev_pixelSensor;


	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, 32, dev_paths1);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths1 + pixelcount;
	int num_paths = dev_path_end - dev_paths1;
	int* rayValid, * rayIndex;
	
	int numRays = num_paths;
	hipMalloc((void**)&rayValid, sizeof(int) * pixelcount);
	hipMalloc((void**)&rayIndex, sizeof(int) * pixelcount);
	
	hipDeviceSynchronize();
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	while (numRays && depth < 32) {

		// clean shading chunks
		depth++;
		hipMemset(dev_intersections1, 0, pixelcount * sizeof(ShadeableIntersection));
		hipMemset(rayValid, 0, sizeof(int) * pixelcount);
		dim3 numblocksPathSegmentTracing = (numRays + blockSize1d - 1) / blockSize1d;
#if !STOCHASTIC_SAMPLING && FIRST_INTERSECTION_CACHING
		if (iter != 1 && depth == 0)
		{
			hipMemcpy(dev_intersections1, dev_intersectionCache, pixelcount * sizeof(ShadeableIntersection), hipMemcpyHostToHost);
			hipMemcpy(dev_paths1, dev_pathCache, pixelcount * sizeof(PathSegment), hipMemcpyHostToHost);
			hipMemcpy(rayValid, dev_rayValidCache, sizeof(int) * pixelcount, hipMemcpyHostToHost);
			addBackground << < numblocksPathSegmentTracing, blockSize1d >> > (dev_image, dev_imageCache, pixelcount);
		}
		if (iter == 1||(iter!=1&&depth>0))
		{
#endif
			// tracing
			if (hst_scene->media.size() == 0)
			{
				compute_intersection_bvh_no_volume << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth
					, numRays
					, dev_paths1
					, dev_sceneInfo
					, dev_intersections1
					, rayValid
					, dev_film
					);
			}
			else
			{
				compute_intersection_bvh_volume_naive << <numblocksPathSegmentTracing, blockSize1d >> > (
					iter
					, depth
					, numRays
					, dev_paths1
					, dev_sceneInfo
					, dev_intersections1
					, rayValid
					, dev_film
					);
			}

#if !STOCHASTIC_SAMPLING && FIRST_INTERSECTION_CACHING
		}
		if (iter == 1 && depth == 0)
		{
			hipMemcpy(dev_intersectionCache, dev_intersections1, pixelcount * sizeof(ShadeableIntersection), hipMemcpyHostToHost);
			hipMemcpy(dev_pathCache, dev_paths1, pixelcount * sizeof(PathSegment), hipMemcpyHostToHost);
			hipMemcpy(dev_rayValidCache, rayValid, sizeof(int) * pixelcount, hipMemcpyHostToHost);
			hipMemcpy(dev_imageCache, dev_image, sizeof(glm::vec3) * pixelcount, hipMemcpyHostToHost);
		}
#endif

		hipDeviceSynchronize();
		checkCUDAError("compute_intersection");

		

#if SORT_BY_MATERIAL_TYPE
		numRays = compact_rays(rayValid, rayIndex, numRays, true);
#else
		numRays = compact_rays(rayValid, rayIndex, numRays);
#endif

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.
		if (!numRays) break;
		dim3 numblocksLightScatter = (numRays + blockSize1d - 1) / blockSize1d;
#if USE_MIS
		scatter_on_intersection_mis << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			numRays,
			dev_intersections1,
			dev_paths1,
			dev_sceneInfo,
			rayValid,
			dev_image
			);
#else
		scatter_on_intersection << <numblocksPathSegmentTracing, blockSize1d , BxDFMaxSize * blockSize1d >> > (
			iter,
			numRays,
			dev_intersections1,
			dev_paths1,
			dev_sceneInfo,
			rayValid,
			dev_film
			);
#endif
		hipDeviceSynchronize();
		checkCUDAError("scatter_on_intersection");

		numRays = compact_rays(rayValid, rayIndex, numRays);

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}


	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_film);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	hipFree(rayValid);
	hipFree(rayIndex);

	checkCUDAError("pathtrace");
}

//void DrawGbuffer(int numIter)
//{
//	if (!USE_BVH) throw;
//
//	const Camera& cam = hst_scene->state.camera;
//	const int pixelcount = cam.resolution.x * cam.resolution.y;
//
//	SceneInfoDev dev_sceneInfo{};
//	dev_sceneInfo.dev_materials = dev_materials;
//	dev_sceneInfo.dev_objs = dev_objs;
//	dev_sceneInfo.objectsSize = hst_scene->objects.size();
//	dev_sceneInfo.modelInfo.dev_triangles = dev_triangles;
//	dev_sceneInfo.modelInfo.dev_vertices = dev_vertices;
//	dev_sceneInfo.modelInfo.dev_normals = dev_normals;
//	dev_sceneInfo.modelInfo.dev_uvs = dev_uvs;
//	dev_sceneInfo.modelInfo.dev_tangents = dev_tangents;
//	dev_sceneInfo.modelInfo.dev_fsigns = dev_fsigns;
//	dev_sceneInfo.dev_primitives = dev_primitives;
//#if USE_BVH
//#if MTBVH
//	dev_sceneInfo.dev_mtbvhArray = dev_mtbvhArray;
//	dev_sceneInfo.bvhDataSize = hst_scene->MTBVHArray.size() / 6;
//#else
//	dev_sceneInfo.dev_bvhArray = dev_bvhArray;
//	dev_sceneInfo.bvhDataSize = hst_scene->bvhTreeSize;
//#endif
//#endif // 
//	dev_sceneInfo.skyboxObj = hst_scene->skyboxTextureObj;
//
//	const dim3 blockSize2d(8, 8);
//	const dim3 blocksPerGrid2d(
//		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
//		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);
//	
//	const int blockSize1d = 128;
//	dim3 numblocksPathSegmentTracing = (pixelcount + blockSize1d - 1) / blockSize1d;
//	SceneGbuffer dev_gbuffer;
//	glm::vec3* dev_albedo,*dev_normal;
//	hipMalloc(&dev_albedo, pixelcount * sizeof(glm::vec3));
//	hipMalloc(&dev_normal, pixelcount * sizeof(glm::vec3));
//	dev_gbuffer.dev_albedo = dev_albedo;
//	dev_gbuffer.dev_normal = dev_normal;
//	for (int i = 0; i < numIter; i++)
//	{
//		generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, i, MAX_DEPTH, dev_paths1);
//		draw_gbuffer << <numblocksPathSegmentTracing, blockSize1d >> > (pixelcount, dev_paths1, dev_sceneInfo, dev_gbuffer);
//	}
//	hst_scene->state.albedo.resize(pixelcount);
//	hst_scene->state.normal.resize(pixelcount);
//	hipMemcpy(hst_scene->state.albedo.data(), dev_albedo, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
//	hipMemcpy(hst_scene->state.normal.data(), dev_normal, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
//	hipFree(dev_albedo);
//	hipFree(dev_normal);
//	for (int i = 0; i < pixelcount; i++)
//	{
//		hst_scene->state.albedo[i] /= (float)numIter;
//		hst_scene->state.normal[i] /= (float)numIter;
//	}
//}
